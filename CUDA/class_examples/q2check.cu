
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <string.h>

int size_n, SEED;

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__global__ void XOR_Sum(int *a, int sz) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int n = sz/2;

	while(n != 0) {
		if(i < n)
			a[i] = (a[i]^a[i+n]);
		__syncthreads();
		if(sz % 2 == 1) {
			a[n] = a[2*n];
			sz = n+1;
		}
		else
			sz = n;
		n = (sz/2);
	}
}


int main() {
	struct timeval start, end, t_start, t_end;
	int *arr, *gpu_arr; 
	int X_sum, blocks; // array whose xor we need to calculate
	
	scanf("%d", &size_n);

	/* Allocate host (CPU) memory and initialize*/
	arr = (int *)malloc(size_n * sizeof(int));
	if(!arr) {
		printf("Cannot declare required memory\n");
		exit(-1);
	}
	// srand(SEED); // srand sets the seed which is used for generating random numbers 

	for(int i = 0;i < size_n;i++)
		scanf("%d",&arr[i]);

	/* Allocate GPU memory and copy from CPU --> GPU*/
	hipMalloc(&gpu_arr, size_n * sizeof(int));
	CUDA_ERROR_EXIT("hipMalloc");

	hipMemcpy(gpu_arr, arr, size_n * sizeof(int) , hipMemcpyHostToDevice);
	CUDA_ERROR_EXIT("hipMemcpy");
	
	blocks = (size_n / 2) / 1024;
	
	if((size_n / 2) % 1024)
	       ++blocks;

	XOR_Sum<<< blocks, 1024>>>(gpu_arr,size_n);
	hipMemcpy(arr, gpu_arr, size_n * sizeof(int) , hipMemcpyDeviceToHost);

	X_sum = arr[0];


	// gettimeofday(&t_start, NULL); // for total time

	// gettimeofday(&start, NULL); // when actual processing starts (call to global)

	// gettimeofday(&end, NULL);
	
	// gettimeofday(&t_end, NULL);

	// printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));


	printf("XOR Sum: %d\n", X_sum);

	return 0;
}