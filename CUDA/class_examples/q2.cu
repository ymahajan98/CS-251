
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <string.h>

int size_n;
unsigned int SEED;

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__global__ void XOR_Sum(int *a, int sz) {
	long long int i = (blockDim.x * blockIdx.x) + threadIdx.x;
	int n = sz/2;
	if(i >= n)
		return;
	a[i] = (a[i] ^ a[i+n]);
	if(sz %2 == 1)
		a[n] = a[2*n];
}


int main(int argc, char **argv) {
	struct timeval start, end, t_start, t_end;
	int *arr, *gpu_arr; 
	int X_sum, blocks; // array whose xor we need to calculate
	
	if(argc != 3) {
		printf("Invalid arguments!!\n");
		exit(-1);
	}
	else {
		size_n = atoi(argv[1]);
		SEED = atoi(argv[2]);
		if(size_n <= 0) {
			printf("Invalid value of number of terms!\n");
			exit(-1);
		}
	}

	/* Allocate host (CPU) memory and initialize*/
	arr = (int *)malloc(size_n * sizeof(int));
	if(!arr) {
		printf("Cannot declare required memory\n");
		exit(-1);
	}
	srand(SEED); // srand sets the seed which is used for generating random numbers 

	for(int i = 0;i < size_n;i++)
		arr[i] = rand();

	int sum = 0;
	for(int i = 0;i < size_n;i++)
		sum = (sum ^ arr[i]);
	printf("Sequential XOR: %d\n",sum);
	gettimeofday(&t_start, NULL); // for total time

	/* Allocate GPU memory and copy from CPU --> GPU*/
	hipMalloc(&gpu_arr, size_n * sizeof(int));
	CUDA_ERROR_EXIT("hipMalloc");

	hipMemcpy(gpu_arr, arr, size_n * sizeof(int) , hipMemcpyHostToDevice);
	CUDA_ERROR_EXIT("hipMemcpy");
	
	gettimeofday(&start, NULL); // when actual processing starts (call to global)

	blocks = (size_n / 2) / 1024;
	
	if((size_n / 2) % 1024)
	       ++blocks;

	int sz = size_n, nn = sz/2;

	while(nn != 0) {
		XOR_Sum<<<blocks, 1024>>> (gpu_arr, sz);
		if(sz%2 == 1) 
			sz = nn + 1;
		else
			sz = nn;
		nn = sz/2;

	}

	gettimeofday(&end, NULL);

	hipMemcpy(arr, gpu_arr, sizeof(int) , hipMemcpyDeviceToHost);
	gettimeofday(&t_end, NULL);

	X_sum = arr[0];
	printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));


	hipFree(gpu_arr);

	printf("XOR Sum: %d\n", X_sum);

	return 0;
}